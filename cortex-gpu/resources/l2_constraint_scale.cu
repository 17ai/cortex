
#include <hip/hip_runtime.h>
//lhs = lhs * rhs;
template<typename dtype>
__device__
void l2_constraint_scale (dtype* l2_squared, dtype l2_max, int count)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if ( i < count ) {
    dtype val = sqrt(l2_squared[i]);
    dtype multiplier = val > l2_max ? (l2_max / val) : 1.0;
    l2_squared[i] = multiplier;
  }
}


extern "C"
__global__
void l2_constraint_scale_d (double* l2_squared, double l2_max, int count)
{
  l2_constraint_scale(l2_squared, l2_max, count);
}


extern "C"
__global__
void l2_constraint_scale_f (float* l2_squared, double l2_max, int count)
{
  l2_constraint_scale(l2_squared, static_cast<float>(l2_max), count);
}
